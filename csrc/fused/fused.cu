#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2024 by SageAttention team.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <ATen/cuda/HIPContext.h>
#include <torch/extension.h>

#include "../dispatch_utils.h"
#include "../utils.cuh"
#include "../reduction_utils.cuh"
#include "../numeric_conversion.cuh"
#include "../cp_async.cuh"
#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>

enum class QuantType
{
  kInt8,
  kInt4,
};

template <typename T>
__device__ __forceinline__ float convert_to_float(T val)
{
  static_assert(std::is_same<T, half>::value || std::is_same<T, hip_bfloat16>::value, "Only half and bfloat16 are supported");

  if constexpr (std::is_same<T, half>::value)
  {
    return __half2float(val);
  }
  else if constexpr (std::is_same<T, hip_bfloat16>::value)
  {
    return __bfloat162float(val);
  }
}

template <typename T>
__device__ __forceinline__ T convert_from_float(float val)
{
  static_assert(std::is_same<T, half>::value || std::is_same<T, hip_bfloat16>::value, "Only half and bfloat16 are supported");

  if constexpr (std::is_same<T, half>::value)
  {
    return __float2half_rn(val);
  }
  else if constexpr (std::is_same<T, hip_bfloat16>::value)
  {
    return __float2bfloat16_rn(val);
  }
}

template <uint32_t head_dim, uint32_t BLOCK_SIZE, uint32_t num_pack_per_thread = 1, bool has_sm_scale = false, bool sub_mean = false, typename T>
__global__ void QuantInt8Kernel(T *__restrict__ input, T *__restrict__ mean, int8_t *__restrict__ output, float *__restrict__ scale, float sm_scale, const uint32_t num_tokens, 
                            const uint32_t stride_bz_input, const uint32_t stride_seq_input, const uint32_t stride_h_input,
                            const uint32_t stride_bz_mean, const uint32_t stride_h_mean,
                            const uint32_t stride_bz_output, const uint32_t stride_seq_output, const uint32_t stride_h_output,
                            const uint32_t stride_bz_scale, const uint32_t stride_h_scale)
{
  static_assert(std::is_same<T, half>::value || std::is_same<T, hip_bfloat16>::value, "Only half and bfloat16 are supported");
  static_assert(num_pack_per_thread > 0, "The number of pack per thread must be greater than 0");

  constexpr uint32_t pack_size = 8; // float4 contains 8 half or 8 bfloat16
  constexpr uint32_t num_threads_per_token = head_dim / pack_size;

  static_assert(num_threads_per_token <= 32, "The number of threads per token must be less than or equal to warp size");

  T x_val[num_pack_per_thread][8];
  T mean_val[8];
  float x_val_float[num_pack_per_thread][8];
  float mean_val_float[8];

  uint32_t bx = blockIdx.x;
  uint32_t head_id = blockIdx.y;
  uint32_t batch_id = blockIdx.z;
  uint32_t thread_id = threadIdx.x;

  uint32_t thread_base_token = bx * BLOCK_SIZE + thread_id / num_threads_per_token;
  T *input_ptr_base = input + batch_id * stride_bz_input + head_id * stride_h_input + thread_base_token * stride_seq_input + thread_id % num_threads_per_token * pack_size;
  T *mean_ptr_base = mean + batch_id * stride_bz_mean + head_id * stride_h_mean + thread_id % num_threads_per_token * pack_size;
  int8_t *output_ptr_base = output + batch_id * stride_bz_output + head_id * stride_h_output + thread_base_token * stride_seq_output + thread_id % num_threads_per_token * pack_size;
  float *scale_ptr_base = scale + batch_id * stride_bz_scale + head_id * stride_h_scale + bx;

  if constexpr (sub_mean)
  {
    *(float4*)(&mean_val[0]) = *(float4*)(mean_ptr_base);
#pragma unroll
    for (uint32_t j = 0; j < 8; j++)
    {
      mean_val_float[j] = convert_to_float(mean_val[j]);
    }
  }

  constexpr uint32_t iter_stride = BLOCK_SIZE / num_pack_per_thread;

  // load the data
  for (uint32_t i = 0; i < num_pack_per_thread; i++)
  {
    if (thread_base_token + i * iter_stride < num_tokens)
    {
      *(float4*)(&x_val[i][0]) = *(float4*)(input_ptr_base + i * iter_stride * stride_seq_input);
#pragma unroll
      for (uint32_t j = 0; j < 8; j++)
      {
        x_val_float[i][j] = convert_to_float(x_val[i][j]);
      }

      if constexpr (sub_mean)
      {
#pragma unroll
        for (uint32_t j = 0; j < 8; j++)
        {
          x_val_float[i][j] -= mean_val_float[j];
        }
      }

      if constexpr (has_sm_scale)
      {
#pragma unroll
        for (uint32_t j = 0; j < 8; j++)
        {
          x_val_float[i][j] *= sm_scale;
        }
      }
    }
    else
    {
#pragma unroll
      for (uint32_t j = 0; j < 8; j++)
      {
        x_val_float[i][j] = 0.0f;
      }
    }
  }

  float amax_val = 0.0000001f; // prevent from dividing by zero

#pragma unroll
  for (uint32_t i = 0; i < num_pack_per_thread; i++)
  {
#pragma unroll
    for (uint32_t j = 0; j < 8; j++)
    {
      amax_val = fmaxf(amax_val, fabsf(x_val_float[i][j]));
    }
  }

  __shared__ float s_amax;
  const float block_amax_val = vllm::blockReduceMax(amax_val);
  if (thread_id == 0)
  {
    s_amax = block_amax_val;
    scale_ptr_base[0] = s_amax / 127.0f;
  }

  __syncthreads();

  float tmp_scale = 127.0f / s_amax;

  char4 o_val[num_pack_per_thread][2];

#pragma unroll
  for (uint32_t i = 0; i < num_pack_per_thread; i++)
  {
#pragma unroll
    for (uint32_t j = 0; j < 2; j += 1)
    {
      o_val[i][j] = make_char4(
        float_to_int8_rn(x_val_float[i][j * 4 + 0] * tmp_scale),
        float_to_int8_rn(x_val_float[i][j * 4 + 1] * tmp_scale),
        float_to_int8_rn(x_val_float[i][j * 4 + 2] * tmp_scale),
        float_to_int8_rn(x_val_float[i][j * 4 + 3] * tmp_scale)
      );
    }
  }

  // int8 result
#pragma unroll
  for (uint32_t i = 0; i < num_pack_per_thread; i++)
  {
    
    if (thread_base_token + i * iter_stride < num_tokens)
    {
      *reinterpret_cast<float2*>(output_ptr_base + i * iter_stride * stride_seq_output) = *reinterpret_cast<float2*>(&o_val[i][0]);
    }
  }
}

template <uint32_t head_dim, uint32_t BLOCK_SIZE, uint32_t num_pack_per_thread = 1, typename T>
__global__ void SubMeanKernel(T *__restrict__ input, T *__restrict__ mean, half *__restrict__ output, const uint32_t num_tokens, 
                            const uint32_t stride_bz_input, const uint32_t stride_seq_input, const uint32_t stride_h_input,
                            const uint32_t stride_bz_mean, const uint32_t stride_h_mean,
                            const uint32_t stride_bz_output, const uint32_t stride_seq_output, const uint32_t stride_h_output)
{
  static_assert(std::is_same<T, half>::value || std::is_same<T, hip_bfloat16>::value, "Only half and bfloat16 are supported");
  static_assert(num_pack_per_thread > 0, "The number of pack per thread must be greater than 0");

  using T2 = typename std::conditional<std::is_same<T, half>::value, half2, nv_bfloat162>::type;

  constexpr uint32_t pack_size = 8; // float4 contains 8 half or 8 bfloat16
  constexpr uint32_t num_threads_per_token = head_dim / pack_size;

  static_assert(num_threads_per_token <= 32, "The number of threads per token must be less than or equal to warp size");

  T2 x_val[num_pack_per_thread][4];
  T2 mean_val[4];

  uint32_t bx = blockIdx.x;
  uint32_t head_id = blockIdx.y;
  uint32_t batch_id = blockIdx.z;
  uint32_t thread_id = threadIdx.x;

  uint32_t thread_base_token = bx * BLOCK_SIZE + thread_id / num_threads_per_token;
  T *input_ptr_base = input + batch_id * stride_bz_input + head_id * stride_h_input + thread_base_token * stride_seq_input + thread_id % num_threads_per_token * pack_size;
  T *mean_ptr_base = mean + batch_id * stride_bz_mean + head_id * stride_h_mean + thread_id % num_threads_per_token * pack_size;
  half *output_ptr_base = output + batch_id * stride_bz_output + head_id * stride_h_output + thread_base_token * stride_seq_output + thread_id % num_threads_per_token * pack_size;

  *(float4*)(&mean_val[0]) = *(float4*)(mean_ptr_base);

  constexpr uint32_t iter_stride = BLOCK_SIZE / num_pack_per_thread;

  // load the data
  for (uint32_t i = 0; i < num_pack_per_thread; i++)
  {
    if (thread_base_token + i * iter_stride < num_tokens)
    {
      *(float4*)(&x_val[i][0]) = *(float4*)(input_ptr_base + i * iter_stride * stride_seq_input);
#pragma unroll
      for (uint32_t j = 0; j < 4; j++)
      {
        x_val[i][j] = __hsub2(x_val[i][j], mean_val[j]);

        if constexpr (std::is_same<T, hip_bfloat16>::value)
        {
          ((half2*)x_val[i])[j] = __float22half2_rn(__bfloat1622float2(x_val[i][j])); 
        }
      }
    }
  }

#pragma unroll
  for (uint32_t i = 0; i < num_pack_per_thread; i++)
  {
    if (thread_base_token + i * iter_stride < num_tokens)
    {
      *reinterpret_cast<float4*>(output_ptr_base + i * iter_stride * stride_seq_output) = *reinterpret_cast<float4*>(&x_val[i][0]);
    }
  }
}

template <uint32_t head_dim, uint32_t CTA_SIZE, bool pad_zero=false, typename T>
__global__ void TransposePadPermuteKernel(T *__restrict__ input, T *__restrict__ output, const uint32_t num_tokens,
                            const uint32_t stride_bz_input, const uint32_t stride_seq_input, const uint32_t stride_h_input,
                            const uint32_t stride_bz_output, const uint32_t stride_d_output, const uint32_t stride_h_output)
{

  static_assert(std::is_same<T, half>::value || std::is_same<T, hip_bfloat16>::value, "Only half and bfloat16 are supported");

  constexpr uint32_t pack_size = 8; // float4 contains 8 half or 8 bfloat16
  uint32_t num_threads_per_token = head_dim / pack_size;
  uint32_t num_threads_per_cta = CTA_SIZE / pack_size;

  uint32_t bx = blockIdx.x;
  uint32_t head_id = blockIdx.y;
  uint32_t batch_id = blockIdx.z;
  uint32_t thread_id = threadIdx.x;

  uint32_t thread_base_token = bx * CTA_SIZE + thread_id / num_threads_per_token;

  T *input_ptr_base = input + batch_id * stride_bz_input + head_id * stride_h_input + thread_base_token * stride_seq_input + thread_id % num_threads_per_token * pack_size;
  T* output_ptr_base = output + batch_id * stride_bz_output + head_id * stride_h_output + bx * CTA_SIZE + thread_id % num_threads_per_cta * pack_size + thread_id / num_threads_per_cta * stride_d_output;

  __shared__ T shared_load[CTA_SIZE][head_dim];
  __shared__ T shared_store[head_dim][CTA_SIZE];

  // 0, 1, 4, 5, 8, 9, 12, 13, 2, 3, 6, 7, 10, 11, 14, 15
  // permute on the seq dimension for fp8 mma
  uint32_t smem_load_row_base = ((thread_id / num_threads_per_token) / 16) * 16;
  uint32_t smem_load_row_mod = (thread_id / num_threads_per_token) % 16;
  uint32_t smem_load_row = smem_load_row_base + (smem_load_row_mod  / 8) * 2 + ((smem_load_row_mod / 2) % 4) * 4 + (smem_load_row_mod % 2);

  constexpr cp_async::SharedMemFillMode fill_mode = pad_zero ? cp_async::SharedMemFillMode::kFillZero : cp_async::SharedMemFillMode::kNoFill;
  cp_async::pred_load_128b<cp_async::PrefetchMode::kNoPrefetch, fill_mode>(shared_load[smem_load_row] + thread_id % num_threads_per_token * pack_size, input_ptr_base, thread_base_token < num_tokens);
  cp_async::commit_group();
  cp_async::wait_group<0>();
  __syncthreads();

  uint32_t smem_row_base = thread_id % CTA_SIZE;
  uint32_t smem_col_base = thread_id / CTA_SIZE;
  uint32_t smem_col_stride = head_dim / 8;

  // TODO: use ldmatrix to do permutation
#pragma unroll
  for (uint32_t i = 0; i < 8; i++)
  {
    shared_store[smem_col_base + i * smem_col_stride][smem_row_base] = shared_load[smem_row_base][smem_col_base + i * smem_col_stride];
  }

  __syncthreads();

  *(float4*)(output_ptr_base) = *(float4*)(&shared_store[thread_id / num_threads_per_cta][thread_id % num_threads_per_cta * pack_size]);
}


template<uint32_t pad_size, bool sub_mean = false, typename T>
__global__ void MeanScaleKernel(T *__restrict__ input, int8_t *__restrict__ output, float *__restrict__ mean, float *__restrict__ scale, const float scale_max, const uint32_t num_tokens,
                            const uint32_t stride_bz_input, const uint32_t stride_d_input, const uint32_t stride_h_input,
                            const uint32_t stride_bz_output, const uint32_t stride_d_output, const uint32_t stride_h_output,
                            const uint32_t stride_bz_mean, const uint32_t stride_h_mean,
                            const uint32_t stride_bz_scale, const uint32_t stride_h_scale)
{
  static_assert(std::is_same<T, half>::value || std::is_same<T, __hip_bfloat16>::value, "Only half and bfloat16 are supported");

  constexpr uint32_t pack_size = 8; // float4 contains 8 half or 8 bfloat16

  uint32_t head_id = blockIdx.x;
  uint32_t batch_id = blockIdx.y;
  uint32_t d_id = blockIdx.z;
  uint32_t thread_id = threadIdx.x;

  uint32_t num_threads = blockDim.x;
  uint32_t gmem_stride = num_threads * pack_size;
  // pad the number of tokens to 16 to deal with fp8 permute in previous kernel
  uint32_t fp8_padded_num_tokens = (num_tokens + 15) / 16 * 16;
  uint32_t num_iters = fp8_padded_num_tokens / gmem_stride + ((fp8_padded_num_tokens % gmem_stride) > thread_id * pack_size);

  T *input_ptr_base = input + batch_id * stride_bz_input + head_id * stride_h_input + d_id * stride_d_input + thread_id * pack_size;
  int8_t *output_ptr_base = output + batch_id * stride_bz_output + head_id * stride_h_output + d_id * stride_d_output + thread_id * pack_size;

  T x_val[8];
  float x_val_float[8];
  uint32_t x_val_fp8[2];

  float max_val = - 1000000.0f;
  float min_val = 1000000.0f;
  float sum_val = 0.0f;

  for (int i = 0; i < num_iters; i++)
  {
    *(float4*)(&x_val[0]) = *(float4*)(input_ptr_base + i * gmem_stride);
#pragma unroll
    for (uint32_t j = 0; j < 8; j++)
    {
      float x_temp = convert_to_float(x_val[j]);
      max_val = fmaxf(max_val, x_temp);
      min_val = fminf(min_val, x_temp);

      if constexpr (sub_mean)
      {
        sum_val += x_temp;
      }
    }
  }

  // reduce
  __shared__ float s_amax_val;
  __shared__ float s_mean_val;

  float block_max_val = vllm::blockReduceMax(max_val);
  float block_min_val = vllm::blockReduceMin(min_val);
  float block_sum_val;

  if constexpr (sub_mean)
  {
    block_sum_val = vllm::blockReduceSum(sum_val);
  }

  if (thread_id == 0)
  {
    s_mean_val = block_sum_val / fp8_padded_num_tokens;

    if constexpr (sub_mean)
    {
      s_amax_val = fmaxf(fabsf(block_max_val - s_mean_val), fabsf(block_min_val - s_mean_val));
      mean[batch_id * stride_bz_mean + head_id * stride_h_mean + d_id] = s_mean_val;
    }
    else
    {
      s_amax_val = fmaxf(fabsf(block_max_val), fabsf(block_min_val));
    }

    scale[batch_id * stride_bz_scale + head_id * stride_h_scale + d_id] = s_amax_val / scale_max;
  }

  __syncthreads();

  float mean_val = s_mean_val;
  float recp_scale = scale_max / s_amax_val;

  // recalculate num_iters to cover all fp8 output tokens to prevent nan in random initialization
  uint32_t padded_num_tokens = (num_tokens + pad_size - 1) / pad_size * pad_size;
  num_iters = padded_num_tokens / gmem_stride + ((padded_num_tokens % gmem_stride) > thread_id * pack_size);

  for (int i = 0; i < num_iters; i++)
  {
    *(float4*)(&x_val[0]) = *(float4*)(input_ptr_base + i * gmem_stride);
#pragma unroll
    for (uint32_t j = 0; j < 8; j++)
    {
      x_val_float[j] = convert_to_float(x_val[j]);
      if constexpr (sub_mean)
      {
        x_val_float[j] = (x_val_float[j] - mean_val) * recp_scale;
      }
      else
      {
        x_val_float[j] *= recp_scale;
      }
    }

    floatx4_to_e4m3x4(x_val_fp8, x_val_float, x_val_float + 2);
    floatx4_to_e4m3x4(x_val_fp8 + 1, x_val_float + 4, x_val_float + 6);

    *(uint2*)(output_ptr_base + i * gmem_stride) = *(uint2*)(&x_val_fp8[0]);
  }
}

void quant_per_block_int8_cuda(
                torch::Tensor input,
                torch::Tensor output,
                torch::Tensor scale,
                float sm_scale,
                int block_size,
                int tensor_layout)
{
  CHECK_CUDA(input);
  CHECK_CUDA(output);
  CHECK_CUDA(scale);
  
  CHECK_DTYPE(output, torch::kInt8);
  CHECK_DTYPE(scale, torch::kFloat);

  CHECK_LASTDIM_CONTIGUOUS(input);
  CHECK_CONTIGUOUS(output);
  CHECK_CONTIGUOUS(scale);

  CHECK_DIMS(input, 4);
  CHECK_DIMS(output, 4);
  CHECK_DIMS(scale, 3);

  const int batch_size = input.size(0);
  const int head_dim = input.size(3);

  int stride_bz_input = input.stride(0);
  int stride_bz_output = output.stride(0);

  int num_tokens, num_heads;
  int stride_seq_input, stride_h_input, stride_seq_output, stride_h_output;

  if (tensor_layout == 0)
  {
    num_tokens = input.size(1);
    num_heads = input.size(2);
    stride_seq_input = input.stride(1);
    stride_h_input = input.stride(2);
    stride_seq_output = output.stride(1);
    stride_h_output = output.stride(2);
  }
  else
  {
    num_tokens = input.size(2);
    num_heads = input.size(1);
    stride_seq_input = input.stride(2);
    stride_h_input = input.stride(1);
    stride_seq_output = output.stride(2);
    stride_h_output = output.stride(1);
  }

  auto input_dtype = input.scalar_type();
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  DISPATCH_PYTORCH_DTYPE_TO_CTYPE_FP16(input_dtype, c_type, {
    DISPATCH_BLOCK_SIZE(block_size, BLOCK_SIZE, {
      DISPATCH_HEAD_DIM(head_dim, HEAD_DIM, {

        CHECK_SHAPE(output, input.size(0), input.size(1), input.size(2), input.size(3));
        CHECK_SHAPE(scale, batch_size, num_heads, (num_tokens + BLOCK_SIZE - 1) / BLOCK_SIZE);

        dim3 grid((num_tokens + BLOCK_SIZE - 1) / BLOCK_SIZE, num_heads, batch_size);

        constexpr int num_pack_per_thread = (BLOCK_SIZE * (HEAD_DIM / 8) + 1023) / 1024;

        dim3 block(BLOCK_SIZE * (HEAD_DIM / 8) / num_pack_per_thread);

        QuantInt8Kernel<HEAD_DIM, BLOCK_SIZE, num_pack_per_thread, true, false, c_type><<<grid, block, 0, stream>>>(
          reinterpret_cast<c_type*>(input.data_ptr()),
          nullptr,
          output.data_ptr<int8_t>(),
          reinterpret_cast<float*>(scale.data_ptr()),
          sm_scale,
          num_tokens,
          stride_bz_input, stride_seq_input, stride_h_input,
          0, 0,
          stride_bz_output, stride_seq_output, stride_h_output,
          scale.stride(0), scale.stride(1)
        );
      });
    });
  });
}

void quant_per_block_int8_cuda(
                torch::Tensor input,
                torch::Tensor output,
                torch::Tensor scale,
                int block_size,
                int tensor_layout)
{
  CHECK_CUDA(input);
  CHECK_CUDA(output);
  CHECK_CUDA(scale);
  
  CHECK_DTYPE(output, torch::kInt8);
  CHECK_DTYPE(scale, torch::kFloat);

  CHECK_LASTDIM_CONTIGUOUS(input);
  CHECK_CONTIGUOUS(output);
  CHECK_CONTIGUOUS(scale);

  CHECK_DIMS(input, 4);
  CHECK_DIMS(output, 4);
  CHECK_DIMS(scale, 3);

  const int batch_size = input.size(0);
  const int head_dim = input.size(3);

  int stride_bz_input = input.stride(0);
  int stride_bz_output = output.stride(0);

  int num_tokens, num_heads;
  int stride_seq_input, stride_h_input, stride_seq_output, stride_h_output;

  if (tensor_layout == 0)
  {
    num_tokens = input.size(1);
    num_heads = input.size(2);
    stride_seq_input = input.stride(1);
    stride_h_input = input.stride(2);
    stride_seq_output = output.stride(1);
    stride_h_output = output.stride(2);
  }
  else
  {
    num_tokens = input.size(2);
    num_heads = input.size(1);
    stride_seq_input = input.stride(2);
    stride_h_input = input.stride(1);
    stride_seq_output = output.stride(2);
    stride_h_output = output.stride(1);
  }

  auto input_dtype = input.scalar_type();
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  DISPATCH_PYTORCH_DTYPE_TO_CTYPE_FP16(input_dtype, c_type, {
    DISPATCH_BLOCK_SIZE(block_size, BLOCK_SIZE, {
      DISPATCH_HEAD_DIM(head_dim, HEAD_DIM, {

        CHECK_SHAPE(output, input.size(0), input.size(1), input.size(2), input.size(3));
        CHECK_SHAPE(scale, batch_size, num_heads, (num_tokens + BLOCK_SIZE - 1) / BLOCK_SIZE);

        dim3 grid((num_tokens + BLOCK_SIZE - 1) / BLOCK_SIZE, num_heads, batch_size);

        constexpr int num_pack_per_thread = (BLOCK_SIZE * (HEAD_DIM / 8) + 1023) / 1024;

        dim3 block(BLOCK_SIZE * (HEAD_DIM / 8) / num_pack_per_thread);

        QuantInt8Kernel<HEAD_DIM, BLOCK_SIZE, num_pack_per_thread, false, false, c_type><<<grid, block, 0, stream>>>(
          reinterpret_cast<c_type*>(input.data_ptr()),
          nullptr,
          output.data_ptr<int8_t>(),
          reinterpret_cast<float*>(scale.data_ptr()),
          0.0f,
          num_tokens,
          stride_bz_input, stride_seq_input, stride_h_input,
          0, 0,
          stride_bz_output, stride_seq_output, stride_h_output,
          scale.stride(0), scale.stride(1)
        );
      });
    });
  });
}

void quant_per_block_int8_fuse_sub_mean_cuda(
                torch::Tensor input,
                torch::Tensor mean,
                torch::Tensor output,
                torch::Tensor scale,
                int block_size,
                int tensor_layout)
{
  CHECK_CUDA(input);
  CHECK_CUDA(mean);
  CHECK_CUDA(output);
  CHECK_CUDA(scale);
  
  CHECK_DTYPE(output, torch::kInt8);
  CHECK_DTYPE(scale, torch::kFloat);

  CHECK_LASTDIM_CONTIGUOUS(input);
  CHECK_CONTIGUOUS(mean);
  CHECK_CONTIGUOUS(output);
  CHECK_CONTIGUOUS(scale);

  CHECK_DIMS(input, 4);
  CHECK_DIMS(mean, 3);
  CHECK_DIMS(output, 4);
  CHECK_DIMS(scale, 3);

  const int batch_size = input.size(0);
  const int head_dim = input.size(3);

  int stride_bz_input = input.stride(0);
  int stride_bz_output = output.stride(0);

  int num_tokens, num_heads;
  int stride_seq_input, stride_h_input, stride_seq_output, stride_h_output;

  if (tensor_layout == 0)
  {
    num_tokens = input.size(1);
    num_heads = input.size(2);
    stride_seq_input = input.stride(1);
    stride_h_input = input.stride(2);
    stride_seq_output = output.stride(1);
    stride_h_output = output.stride(2);
  }
  else
  {
    num_tokens = input.size(2);
    num_heads = input.size(1);
    stride_seq_input = input.stride(2);
    stride_h_input = input.stride(1);
    stride_seq_output = output.stride(2);
    stride_h_output = output.stride(1);
  }

  auto input_dtype = input.scalar_type();
  auto mean_dtype = mean.scalar_type();
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  TORCH_CHECK(input_dtype == mean_dtype, "Input and mean must have the same data type");

  DISPATCH_PYTORCH_DTYPE_TO_CTYPE_FP16(input_dtype, c_type, {
    DISPATCH_BLOCK_SIZE(block_size, BLOCK_SIZE, {
      DISPATCH_HEAD_DIM(head_dim, HEAD_DIM, {

        CHECK_SHAPE(mean, batch_size, num_heads, head_dim);
        CHECK_SHAPE(output, input.size(0), input.size(1), input.size(2), input.size(3));
        CHECK_SHAPE(scale, batch_size, num_heads, (num_tokens + BLOCK_SIZE - 1) / BLOCK_SIZE);

        dim3 grid((num_tokens + BLOCK_SIZE - 1) / BLOCK_SIZE, num_heads, batch_size);

        constexpr int num_pack_per_thread = (BLOCK_SIZE * (HEAD_DIM / 8) + 1023) / 1024;

        dim3 block(BLOCK_SIZE * (HEAD_DIM / 8) / num_pack_per_thread);

        QuantInt8Kernel<HEAD_DIM, BLOCK_SIZE, num_pack_per_thread, false, true, c_type><<<grid, block, 0, stream>>>(
          reinterpret_cast<c_type*>(input.data_ptr()),
          reinterpret_cast<c_type*>(mean.data_ptr()),
          output.data_ptr<int8_t>(),
          reinterpret_cast<float*>(scale.data_ptr()),
          0.0f,
          num_tokens,
          stride_bz_input, stride_seq_input, stride_h_input,
          mean.stride(0), mean.stride(1),
          stride_bz_output, stride_seq_output, stride_h_output,
          scale.stride(0), scale.stride(1)
        );
      });
    });
  });
}

// use block size 128 and warp_block size 32
void quant_per_warp_int8_cuda(
                torch::Tensor input,
                torch::Tensor output,
                torch::Tensor scale,
                int block_size,
                int warp_block_size,
                int tensor_layout)
{
  CHECK_CUDA(input);
  CHECK_CUDA(output);
  CHECK_CUDA(scale);
  
  CHECK_DTYPE(output, torch::kInt8);
  CHECK_DTYPE(scale, torch::kFloat);

  CHECK_LASTDIM_CONTIGUOUS(input);
  CHECK_CONTIGUOUS(output);
  CHECK_CONTIGUOUS(scale);

  CHECK_DIMS(input, 4);
  CHECK_DIMS(output, 4);
  CHECK_DIMS(scale, 3);

  const int batch_size = input.size(0);
  const int head_dim = input.size(3);

  int stride_bz_input = input.stride(0);
  int stride_bz_output = output.stride(0);

  int num_tokens, num_heads;
  int stride_seq_input, stride_h_input, stride_seq_output, stride_h_output;

  if (tensor_layout == 0)
  {
    num_tokens = input.size(1);
    num_heads = input.size(2);
    stride_seq_input = input.stride(1);
    stride_h_input = input.stride(2);
    stride_seq_output = output.stride(1);
    stride_h_output = output.stride(2);
  }
  else
  {
    num_tokens = input.size(2);
    num_heads = input.size(1);
    stride_seq_input = input.stride(2);
    stride_h_input = input.stride(1);
    stride_seq_output = output.stride(2);
    stride_h_output = output.stride(1);
  }

  auto input_dtype = input.scalar_type();
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  DISPATCH_PYTORCH_DTYPE_TO_CTYPE_FP16(input_dtype, c_type, {
    DISPATCH_BLOCK_SIZE(block_size, BLOCK_SIZE, {
      DISPATCH_WARP_BLOCK_SIZE(warp_block_size, WARP_BLOCK_SIZE, {
        DISPATCH_HEAD_DIM(head_dim, HEAD_DIM, {

          CHECK_SHAPE(output, input.size(0), input.size(1), input.size(2), input.size(3));
          CHECK_SHAPE(scale, batch_size, num_heads, (num_tokens + BLOCK_SIZE - 1) / BLOCK_SIZE * (BLOCK_SIZE / WARP_BLOCK_SIZE));

          dim3 grid((num_tokens + BLOCK_SIZE - 1) / BLOCK_SIZE * (BLOCK_SIZE / WARP_BLOCK_SIZE), num_heads, batch_size);

          constexpr int num_pack_per_thread = (WARP_BLOCK_SIZE * (HEAD_DIM / 8) + 1023) / 1024;

          dim3 block(WARP_BLOCK_SIZE * (HEAD_DIM / 8) / num_pack_per_thread);

          QuantInt8Kernel<HEAD_DIM, WARP_BLOCK_SIZE, num_pack_per_thread, false, false, c_type><<<grid, block, 0, stream>>>(
            reinterpret_cast<c_type*>(input.data_ptr()),
            nullptr,
            output.data_ptr<int8_t>(),
            reinterpret_cast<float*>(scale.data_ptr()),
            0.0,
            num_tokens,
            stride_bz_input, stride_seq_input, stride_h_input,
            0, 0,
            stride_bz_output, stride_seq_output, stride_h_output,
            scale.stride(0), scale.stride(1)
          );
        });
      });
    });
  });
}

void sub_mean_cuda(
                torch::Tensor input,
                torch::Tensor mean,
                torch::Tensor output,
                int tensor_layout)
{
  CHECK_CUDA(input);
  CHECK_CUDA(mean);
  CHECK_CUDA(output);
  
  CHECK_LASTDIM_CONTIGUOUS(input);
  CHECK_CONTIGUOUS(mean);
  CHECK_CONTIGUOUS(output);

  CHECK_DIMS(input, 4);
  CHECK_DIMS(mean, 3);
  CHECK_DIMS(output, 4);

  CHECK_DTYPE(output, torch::kHalf);

  const int batch_size = input.size(0);
  const int head_dim = input.size(3);

  int stride_bz_input = input.stride(0);
  int stride_bz_output = output.stride(0);

  int num_tokens, num_heads;
  int stride_seq_input, stride_h_input, stride_seq_output, stride_h_output;

  if (tensor_layout == 0)
  {
    num_tokens = input.size(1);
    num_heads = input.size(2);
    stride_seq_input = input.stride(1);
    stride_h_input = input.stride(2);
    stride_seq_output = output.stride(1);
    stride_h_output = output.stride(2);
  }
  else
  {
    num_tokens = input.size(2);
    num_heads = input.size(1);
    stride_seq_input = input.stride(2);
    stride_h_input = input.stride(1);
    stride_seq_output = output.stride(2);
    stride_h_output = output.stride(1);
  }

  auto input_dtype = input.scalar_type();
  auto mean_dtype = mean.scalar_type();
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  TORCH_CHECK(input_dtype == mean_dtype, "Input and mean must have the same data type");

  DISPATCH_PYTORCH_DTYPE_TO_CTYPE_FP16(input_dtype, c_type, {
    DISPATCH_HEAD_DIM(head_dim, HEAD_DIM, {
        
        CHECK_SHAPE(mean, batch_size, num_heads, head_dim);
        CHECK_SHAPE(output, input.size(0), input.size(1), input.size(2), input.size(3));
  
        constexpr int BLOCK_SIZE = (HEAD_DIM == 128) ? 64 : 128;

        dim3 grid((num_tokens + BLOCK_SIZE - 1) / BLOCK_SIZE, num_heads, batch_size);

        constexpr int num_pack_per_thread = (BLOCK_SIZE * (HEAD_DIM / 8) + 1023) / 1024;

        dim3 block(BLOCK_SIZE * (HEAD_DIM / 8) / num_pack_per_thread);

        SubMeanKernel<HEAD_DIM, BLOCK_SIZE, num_pack_per_thread><<<grid, block, 0, stream>>>(
          reinterpret_cast<c_type*>(input.data_ptr()),
          reinterpret_cast<c_type*>(mean.data_ptr()),
          reinterpret_cast<half*>(output.data_ptr()),
          num_tokens,
          stride_bz_input, stride_seq_input, stride_h_input,
          mean.stride(0), mean.stride(1),
          stride_bz_output, stride_seq_output, stride_h_output
        );
    });
  });
}

void transpose_pad_permute_cuda(
                torch::Tensor input,
                torch::Tensor output,
                int tensor_layout)
{
  CHECK_CUDA(input);
  CHECK_CUDA(output);

  CHECK_LASTDIM_CONTIGUOUS(input);
  CHECK_CONTIGUOUS(output);

  CHECK_DIMS(input, 4);
  CHECK_DIMS(output, 4);

  constexpr int CTA_SIZE = 64;

  const int batch_size = input.size(0);
  const int head_dim = input.size(3);

  int stride_bz_input = input.stride(0);
  int stride_bz_output = output.stride(0);

  int num_tokens, padded_num_tokens, num_heads;
  int stride_seq_input, stride_h_input, stride_d_output, stride_h_output;

  if (tensor_layout == 0)
  {
    num_tokens = input.size(1);
    num_heads = input.size(2);
    stride_seq_input = input.stride(1);
    stride_h_input = input.stride(2);
    stride_d_output = output.stride(1);
    stride_h_output = output.stride(2);

    padded_num_tokens = (num_tokens + CTA_SIZE - 1) / CTA_SIZE * CTA_SIZE;

    CHECK_SHAPE(output, batch_size, head_dim, num_heads, padded_num_tokens);
  }
  else
  {
    num_tokens = input.size(2);
    num_heads = input.size(1);
    stride_seq_input = input.stride(2);
    stride_h_input = input.stride(1);
    stride_d_output = output.stride(2);
    stride_h_output = output.stride(1);

    padded_num_tokens = (num_tokens + CTA_SIZE - 1) / CTA_SIZE * CTA_SIZE;
    CHECK_SHAPE(output, batch_size, num_heads, head_dim, padded_num_tokens);
  }

  auto input_dtype = input.scalar_type();
  auto output_dtype = output.scalar_type();
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  TORCH_CHECK(input_dtype == output_dtype, "Input and output must have the same data type");

  DISPATCH_PYTORCH_DTYPE_TO_CTYPE_FP16(input_dtype, c_type, {
    DISPATCH_HEAD_DIM(head_dim, HEAD_DIM, {
      dim3 grid(padded_num_tokens / CTA_SIZE, num_heads, batch_size);

      static_assert(CTA_SIZE * HEAD_DIM <= 8192);

      dim3 block(CTA_SIZE * (HEAD_DIM / 8));

      TransposePadPermuteKernel<HEAD_DIM, CTA_SIZE, true, c_type><<<grid, block, 0, stream>>>(
        reinterpret_cast<c_type*>(input.data_ptr()),
        reinterpret_cast<c_type*>(output.data_ptr()),
        num_tokens,
        stride_bz_input, stride_seq_input, stride_h_input,
        stride_bz_output, stride_d_output, stride_h_output
      );
    });
  });
}

void scale_fuse_quant_cuda(
                torch::Tensor input,
                torch::Tensor output,
                torch::Tensor scale,
                int num_tokens,
                float scale_max,
                int tensor_layout)
{
  CHECK_CUDA(input);
  CHECK_CUDA(output);
  CHECK_CUDA(scale);

  // CHECK_DTYPE(output, torch::kInt8);
  CHECK_DTYPE(scale, torch::kFloat);

  CHECK_CONTIGUOUS(input);
  CHECK_CONTIGUOUS(output);
  CHECK_CONTIGUOUS(scale);

  CHECK_DIMS(input, 4);
  CHECK_DIMS(output, 4);
  CHECK_DIMS(scale, 3);

  const int batch_size = input.size(0);
  const int num_tokens_padded = input.size(3);

  int stride_bz_input = input.stride(0);
  int stride_bz_output = output.stride(0);

  int num_heads, head_dim;
  int stride_d_input, stride_h_input, stride_d_output, stride_h_output;

  if (tensor_layout == 0)
  {
    num_heads = input.size(2);
    head_dim = input.size(1);
    stride_d_input = input.stride(1);
    stride_h_input = input.stride(2);
    stride_d_output = output.stride(1);
    stride_h_output = output.stride(2);
  }
  else
  {
    num_heads = input.size(1);
    head_dim = input.size(2);
    stride_d_input = input.stride(2);
    stride_h_input = input.stride(1);
    stride_d_output = output.stride(2);
    stride_h_output = output.stride(1);
  }

  CHECK_SHAPE(output, input.size(0), input.size(1), input.size(2), input.size(3));
  CHECK_SHAPE(scale, batch_size, num_heads, head_dim);

  constexpr int CTA_SIZE = 256;

  dim3 grid(num_heads, batch_size, head_dim);
  dim3 block(CTA_SIZE);

  auto input_dtype = input.scalar_type();
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  DISPATCH_PYTORCH_DTYPE_TO_CTYPE_FP16(input_dtype, c_type, {
    MeanScaleKernel<64, false, c_type><<<grid, block, 0, stream>>>(
      reinterpret_cast<c_type*>(input.data_ptr()),
      reinterpret_cast<int8_t*>(output.data_ptr()),
      nullptr,
      reinterpret_cast<float*>(scale.data_ptr()),
      scale_max,
      num_tokens,
      stride_bz_input, stride_d_input, stride_h_input,
      stride_bz_output, stride_d_output, stride_h_output,
      0, 0,
      scale.stride(0), scale.stride(1)
    );
  });
}

void mean_scale_fuse_quant_cuda(
                torch::Tensor input,
                torch::Tensor output,
                torch::Tensor mean,
                torch::Tensor scale,
                int num_tokens,
                float scale_max,
                int tensor_layout)
{
  CHECK_CUDA(input);
  CHECK_CUDA(output);
  CHECK_CUDA(mean);
  CHECK_CUDA(scale);

  // CHECK_DTYPE(output, torch::kInt8);
  CHECK_DTYPE(mean, torch::kFloat);
  CHECK_DTYPE(scale, torch::kFloat);

  CHECK_CONTIGUOUS(input);
  CHECK_CONTIGUOUS(output);
  CHECK_CONTIGUOUS(mean);
  CHECK_CONTIGUOUS(scale);

  CHECK_DIMS(input, 4);
  CHECK_DIMS(output, 4);
  CHECK_DIMS(mean, 3);
  CHECK_DIMS(scale, 3);

  const int batch_size = input.size(0);
  const int num_tokens_padded = input.size(3);

  int stride_bz_input = input.stride(0);
  int stride_bz_output = output.stride(0);

  int num_heads, head_dim;
  int stride_d_input, stride_h_input, stride_d_output, stride_h_output;

  if (tensor_layout == 0)
  {
    num_heads = input.size(2);
    head_dim = input.size(1);
    stride_d_input = input.stride(1);
    stride_h_input = input.stride(2);
    stride_d_output = output.stride(1);
    stride_h_output = output.stride(2);
  }
  else
  {
    num_heads = input.size(1);
    head_dim = input.size(2);
    stride_d_input = input.stride(2);
    stride_h_input = input.stride(1);
    stride_d_output = output.stride(2);
    stride_h_output = output.stride(1);
  }

  CHECK_SHAPE(output, input.size(0), input.size(1), input.size(2), input.size(3));
  CHECK_SHAPE(mean, batch_size, num_heads, head_dim);
  CHECK_SHAPE(scale, batch_size, num_heads, head_dim);

  constexpr int CTA_SIZE = 256;

  dim3 grid(num_heads, batch_size, head_dim);
  dim3 block(CTA_SIZE);

  auto input_dtype = input.scalar_type();
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  DISPATCH_PYTORCH_DTYPE_TO_CTYPE_FP16(input_dtype, c_type, {
    MeanScaleKernel<64, true, c_type><<<grid, block, 0, stream>>>(
      reinterpret_cast<c_type*>(input.data_ptr()),
      reinterpret_cast<int8_t*>(output.data_ptr()),
      reinterpret_cast<float*>(mean.data_ptr()),
      reinterpret_cast<float*>(scale.data_ptr()),
      scale_max,
      num_tokens,
      stride_bz_input, stride_d_input, stride_h_input,
      stride_bz_output, stride_d_output, stride_h_output,
      mean.stride(0), mean.stride(1),
      scale.stride(0), scale.stride(1)
    );
  });
}